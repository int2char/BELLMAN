#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
};
void parallelor::dellocate(){
};
void parallelor::allocate(int maxn,int maxedge){
}
void parallelor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void parallelor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	has=new int[nodenum*LY*YE];
	p=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m=new int;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=INT_MAX/2,p[i]=-1,has[i]=-1;
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	cout<<cc<<" "<<edges.size()<<endl;
	for(int k=0;k<LY;k++)
	{
		int boff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			int soff=i*nodenum;
			for(int j=0;j<stpair.size();j++)
				{d[boff+soff+stpair[i].first]=0;
				 has[boff+soff+stpair[i].first]=0;
				}
		}
	}
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_p,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_has,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_has,has,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,p,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;
};
parallelor::parallelor()
{
};
__global__ void bellmanhigh(int *st,int *te,int *d,int *has,int *w,int E,int N,int size,int *m,int round)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid],weight=w[eid];
	if(weight<0)return;
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(has[s+off]<round-1)return;
	if(d[s+off]+weight<d[t+off])
		{
			d[t+off]=weight+d[s+off];
			has[t+off]=round;
			*m=1;
		}
}
__global__ void color(int *st,int *te,int *d,int *pre,int *has,int *w,int E,int N,int size,int round)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid],weight=w[eid];
	if(weight<0)return;
	int off=(i/(E*LY))*N+(eid/E)*N*YE;
	if(has[s+off]<round-1)return;
	if(d[s+off]+weight==d[t+off])
		pre[t+off]=s+off;
}
vector<vector<int>> parallelor::routalg(int s,int t,int bw)
{
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	cout<<"size is: "<<size<<endl;
	*m=1;
	int round=1;
	while(*m==1)
	{
		*m=0;
		hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
		bellmanhigh<<<size/1024+1,1024>>>(dev_st,dev_te,dev_d,dev_has,dev_w,edges.size(),nodenum,size,dev_m,round);
	 	color<<<size/1024+1,1024>>>(dev_st,dev_te,dev_d,dev_p,dev_has,dev_w,edges.size(),nodenum,size,round);
		round++;
		hipMemcpy(m,dev_m,sizeof(int),hipMemcpyDeviceToHost);
	}
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	hipFree(dev_w);
	cout<<"before return"<<endl;
	return result;
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}