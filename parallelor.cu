#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	
};
void parallelor::dellocate(){
};
void parallelor::allocate(int maxn,int maxedge){
}
void parallelor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	//sort(tmp.begin(),tmp.end(),pairless());
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void parallelor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	//cout<<"in cuda init"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	stp=stpair;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[edges.size()*LY];
	te=new int[edges.size()*LY];
	d=new int[nodenum*LY*YE];
	w=new int[edges.size()*LY];
	m=new int;
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neie[s].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]<0)
					te[count]=i;
				else
					te[count]=neibn[i][j];
				count++;
			}
	//cout<<"good so far "<<endl;
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=INT_MAX/2;
	int cc=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			w[cc++]=esigns[k][i];
	cout<<cc<<" "<<edges.size()<<endl;
	for(int k=0;k<LY;k++)
	{
		int boff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			int soff=i*nodenum;
			for(int j=0;j<stpair.size();j++)
				d[boff+soff+stpair[i].first]=0;
		}
	}
	//for(int i=0;i<edges.size();i++)
		//cout<<st[i]<<" "<<te[i]<<" "<<w[i]<<endl;
	//for(int i=0;i<nodenum;i++)
		//cout<<d[i]<<endl;
	//cout<<"good so far "<<endl;
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_w,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	if(dev_d==NULL) {
		printf("couldn't allocate %d int's.\n");
	}
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_w,w,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	cout<<nodenum<<endl;
};
parallelor::parallelor()
{
};
__global__ void bellmanhigh(int *st,int *te,int *d,int *w,int E,int N,int size,int*m)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>size)return;	
	int eid=(i%(E*LY));
	int s=st[eid],t=te[eid],weight=w[eid];
	if(weight<0)return;
	int ye=i/(E*LY);
	int ly=eid/E;
	int off=ye*N+ly*N*YE;
	if(d[s+off]+weight<d[t+off])
		{
			d[t+off]=weight+d[s+off];
			*m=1;
		}
}
vector<vector<int>> parallelor::routalg(int s,int t,int bw)
{
	//cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	cout<<"size is: "<<size<<endl;
	*m=1;
	while(*m==1)
	{
		*m=0;
		hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
		bellmanhigh<<<size/512+1,512>>>(dev_st,dev_te,dev_d,dev_w,edges.size(),nodenum,size,dev_m);
		hipMemcpy(m,dev_m,sizeof(int),hipMemcpyDeviceToHost);
	}
	hipMemcpy(d,dev_d,LY*YE*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=0;i<LY*YE*nodenum;i++)
		cout<<d[i]<<" ";*/
	cout<<endl;
	hipStreamSynchronize(0);
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	vector<vector<int>>result(LY,vector<int>());
	hipFree(dev_te);
	hipFree(dev_st);
	hipFree(dev_d);
	hipFree(dev_w);
	cout<<"before return"<<endl;
	return result;
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}