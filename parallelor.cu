#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	
};
void parallelor::dellocate(){
};
void parallelor::allocate(int maxn,int maxedge){
}
void parallelor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	sort(tmp.begin(),tmp.end(),pairless());
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void parallelor::init(pair<vector<edge>,vector<vector<int>>>ext,vector<pair<int,int>>stpair,vector<vector<int>>&relate,ginfo ginf)
{
	cout<<"in cuda init"<<endl;
	nodenum=ginf.pnodesize;
	edges=ext.first;
	vector<vector<int>>esigns;
	esigns=ext.second;
	mark=new int;
	*mark=0;
	W=WD+1;
	st=new int[2*edges.size()*LY];
	te=new int[2*edges.size()*LY];
	d=new int[nodenum*LY*YE];
	esignes=new int[edges.size()*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	neibn=nein;
	vector<vector<int>>neie(nodenum,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			neibn[s].push_back(t);
			neibn[t].push_back(s);
			neie[s].push_back(i);
			neie[t].push_back(i);
		}
	int count=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<nodenum;i++)
			for(int j=0;j<neibn[i].size();j++)
			{
				st[count]=i;
				if(esigns[k][neie[i][j]]==-1)
					te[count]=i;
				te[count]=neibn[i][j];
				count++;
			}
	for(int i=0;i<nodenum*LY*YE;i++)
		d[i]=INT_MAX/2;
	/*for(int k=0;k<LY;k++)
	{
		int boff=k*YE*nodenum;
		for(int i=0;i<YE;i++)
		{
			int soff=i*nodenum;
			for(int j=0;j<stpair.size();j++)
				d[boff+soff+stpair[i].first]=0;
		}
	}*/
	for(int i=0;i<10;i++)
		cout<<d[i]<<endl;
	//hipMalloc((void**)&dev_st,2*LY*edges.size()*sizeof(int));
	//hipMalloc((void**)&dev_te,2*LYedges.size()*sizeof(int));
	//hipMalloc((void**)&dev_d,YE*LY*nodenum*sizeof(int));
	//hipMemcpy(dev_te,te,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_st,st,2*LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	//hipMemcpy(dev_d,d,YE*LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	cout<<"get out"<<endl;
	cout<<nodenum<<endl;
};
parallelor::parallelor()
{

};
__global__ void BFSfast(int *st,int *te,int *d,int round,int E,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int ye=i/(E*LY);
	int eid=(i%(E*LY));
	int ly=eid/E;
	int off=ye*N+ly*N*YE;
	int s=st[eid],t=te[eid];
	if(d[s+off]==round-1&&d[t+off]>round)
		d[t+off]=round;
}
vector<int> parallelor:: routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int kk=1;
	time_t start,end;
	start=clock();
	int size=edges.size()*LY*YE;
	for(int i=0;i<=WD;i++)
		BFSfast<<<size/512+1,512>>>(dev_st,dev_te,dev_d,i,edges.size(),nodenum);
	hipMemcpy(d,dev_d,YE*LY*nodenum*sizeof(int),hipMemcpyDeviceToHost);
	
	end=clock();
	cout<<"GPU time is : "<<end-start<<endl;
	cout<<"over!"<<endl;
	return vector<int>();
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}